#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <fstream>
#include <random>
#include <ctime>
#include <unordered_map>
#include <hiprand/hiprand_kernel.h>
#include <unordered_map>
#include <utility>
#define compare_swap(i,j) if(i > j){ temp = i; i = j; j = temp; }
#define imin(i, j) (i > j ? j : i)

struct pair_hash
{
	template <class T1, class T2>
	std::size_t operator() (const std::pair<T1, T2> &pair) const
	{
		std::size_t other = std::hash<T2>()(pair.second);
		return std::hash<T1>()(pair.first) + 0x9e3779b9 + (other<<6) + (other>>2);
	}
};


inline __device__ int find_num_of_threads(int n, int device_max_thread_per_block){
    if(n > device_max_thread_per_block) return device_max_thread_per_block;
    int i = (n + 31) / 32;
    return i * 32;
}

struct move_struct{
    uint32_t src_start;
    uint32_t src_end;
    uint32_t dst_start;
};

struct gain_struct_t{
    float gain;
    uint32_t i;
    uint32_t j;
};

__device__ float distance_ij(uint32_t city_i, uint32_t city_j, int32_t* coords, uint32_t n){
    float dx = coords[city_i] - coords[city_j];
    float dy = coords[city_i + n] - coords[city_j + n];
    return sqrtf((dx * dx) + (dy * dy));
}


__global__ void apply_2opt_move(int32_t *tour, gain_struct_t *buf, uint32_t n){
    __shared__ gain_struct_t best_gain;
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(threadIdx.x == 0){
        best_gain = buf[0];
    }
    __syncthreads();

    //copy to local
    uint32_t i = best_gain.i;
    uint32_t j = best_gain.j;
    float gain = best_gain.gain;

    if(gain > 0){
        int left = i + id + 1;
        int right = j - id;
        int left_end = (i + j) / 2;
        while(left <= left_end){
            int32_t temp = tour[left];
            int32_t temp2 = tour[left + n];
            tour[left] = tour[right];
            tour[right] = temp;
            tour[left + n] = tour[right + n];
            tour[right + n] = temp2;
            left += 32;
            right -= 32;
        }
    }
}



__global__ void two_opt(int32_t *tour, gain_struct_t *d_inversions, uint32_t n){
    extern __shared__ int32_t s_tour[];
    
    for(int i = threadIdx.x; i < 2 * n; i += blockDim.x){
        s_tour[i] = tour[i];
    }
    __syncthreads();

    int total = ((n - 1) * (n - 2)) / 2;
    
    //int taskPerThread = (total + blockDim.x - 1) / blockDim.x;
    //old this assumes only execute one block,

    int taskPerThread = (total + (gridDim.x * blockDim.x) - 1) / (gridDim.x * blockDim.x); 
    int tid = threadIdx.x + (blockIdx.x * blockDim.x);
    int start = tid * taskPerThread; //including
    int stop = start + taskPerThread; //not including
    if(stop > total - 1) stop = total - 1; 

    int at = start;

    int city_j, city_i, city_i1, city_j1;  
    {
        int b = 3 -  (2 * (int)n);
        int c = 2 * at;
        float delta = (b * b)  - (4 * c);
        float root = (-b - sqrtf(delta)) / 2;
        city_i = (int)(truncf(root));
        float res = (float)city_i * ( ( ((float)city_i) / 2 ) + 1.5 - n) + at;
        city_j = (int)(truncf(res)) + city_i + 2;
    }
    
    gain_struct_t local_best;
    local_best.i = 0;
    local_best.j = 0;
    local_best.gain = 0;

    while(at < stop){
        city_i1 = city_i + 1;
        city_j1 = city_j + 1;
        if(city_j1 == n) city_j1 = 0;        
        float distance_f = 0 - distance_ij(city_i, city_j, s_tour, n) - distance_ij(city_i1, city_j1, s_tour, n) +
        distance_ij(city_i, city_i1, s_tour, n) + distance_ij(city_j, city_j1, s_tour, n);
        if(distance_f > local_best.gain){
            local_best.gain = distance_f;
            local_best.i = city_i;
            local_best.j = city_j;        
        }
        ++at;
        city_j++;
        if(city_j == n){
            city_i++;
            city_j = city_i + 2;
        }
    }
    //write values to global memory
    d_inversions[tid].gain = local_best.gain;
    d_inversions[tid].i = local_best.i;
    d_inversions[tid].j = local_best.j;
}


__global__ void setup_kernel(hiprandState *state){
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    /* Each thread gets same seed, a different sequence number, no offset */    
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void generate_kernel(hiprandState *state, uint32_t n, uint32_t *perturbation_indices, uint32_t iteration_count){
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    int oldid = id;
    int pos = 4 * id;
    /* Copy state to local memory for efficiency */    
    hiprandState localState = state[id];
    while(id < iteration_count){
   	 int indices[4];
   	 do{
       	 	indices[0] = hiprand_uniform(&localState) * n; //likelihood of seeing n is very low, we can say it is btween 0 - n-1
        	indices[1] = hiprand_uniform(&localState) * n; 
        	indices[2] = hiprand_uniform(&localState) * n; 
        	indices[3] = hiprand_uniform(&localState) * n; 
        	int temp;
		compare_swap(indices[0], indices[1]); //network sort
        	compare_swap(indices[2], indices[3]);
        	compare_swap(indices[0], indices[3]);
        	compare_swap(indices[1], indices[2]);
        	compare_swap(indices[0], indices[1]);
        	compare_swap(indices[2], indices[3]);
    	}while(indices[1] <= indices[0] + 1 || indices[2] <= indices[1] + 1 || indices[3] <= indices[2] + 1 || indices[3] + 1 >= n || 
        	(indices[3] == n - 1 && indices[0] == 0));
    	perturbation_indices[pos + 0] = indices[0];
   	perturbation_indices[pos + 1] = indices[1];
    	perturbation_indices[pos + 2] = indices[2];
    	perturbation_indices[pos + 3] = indices[3];
	id += blockDim.x;
	pos = 4*id;
    }
    /* Copy state back to global memory */
    state[oldid] = localState;
}

__global__ void perturbate(int32_t *current_tour, int32_t *other_tour, uint32_t g_i, uint32_t g_j,
    uint32_t g_k, uint32_t g_l, uint32_t g_n){
    uint32_t p_i, p_j, p_k, p_l, n;
   
    p_i = g_i;
    p_j = g_j;
    p_k = g_k;
    p_l = g_l;
    n = g_n;

    move_struct ms[5];
    ms[0].src_start = 0;
    ms[0].src_end = p_i;
    ms[0].dst_start = 0;

    ms[1].src_start = p_k + 1;
    ms[1].src_end = p_l;
    ms[1].dst_start = p_i + 1;
    
    ms[2].src_start = p_j + 1;
    ms[2].src_end = p_k;
    ms[2].dst_start = p_i + p_l - p_k + 1;
    
    ms[3].src_start = p_i + 1;
    ms[3].src_end = p_j;
    ms[3].dst_start = p_i + p_l - p_j + 1;
    
    ms[4].src_start = p_l + 1;
    ms[4].src_end = n - 1;
    ms[4].dst_start = p_l + 1;


    const uint32_t dst_start = ms[blockIdx.x].dst_start;
    const uint32_t src_start = ms[blockIdx.x].src_start;
    const uint32_t src_end = ms[blockIdx.x].src_end;
  
    uint32_t step = threadIdx.x;

    while(src_start + step <= src_end){
        other_tour[dst_start + step] = current_tour[src_start + step];
        other_tour[dst_start + step + n] = current_tour[src_start + step + n];
        step += warpSize;
    }
}
__device__ float calculate_perturbate_cost(uint32_t n, uint32_t g_i, uint32_t g_j, uint32_t g_k, uint32_t g_l,
    int32_t *tour){
    float cost = 0;
    int indices[16];    
    indices[0] = g_i;
    indices[1] = g_i + 1;
    indices[2] = g_j;
    indices[3] = g_j + 1;
    indices[4] = g_k;
    indices[5] = g_k + 1;
    indices[6] = g_l;
    indices[7] = g_l + 1;
    indices[8] = g_i;
    indices[9] = g_k + 1;
    indices[10] = g_l;
    indices[11] = g_j + 1;
    indices[12] = g_k;
    indices[13] = g_i + 1;
    indices[14] = g_j;
    indices[15] = g_l + 1;

    for(int i = 0; i < 8; ++i){
        float distance = distance_ij(indices[2 * i], indices[2 * i + 1], tour, n);
        if(i > 3) cost += distance;
        else cost -= distance;
    }

    return cost;

}

__global__ void reduce(gain_struct_t *idata, gain_struct_t *odata, int len){

    extern __shared__ gain_struct_t sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + (blockIdx.x * (blockDim.x * 2)); //1 thread will load max of 2 values
    
    if(i < len){
        if(i + blockDim.x < len){
            float val1 = idata[i].gain;
            float val2 = idata[i + blockDim.x].gain;
            if(val1 > val2){
                sdata[tid].i = idata[i].i;
                sdata[tid].j = idata[i].j;
                sdata[tid].gain = val1;
            }else{
                sdata[tid].i = idata[i + blockDim.x].i;
                sdata[tid].j = idata[i + blockDim.x].j;
                sdata[tid].gain = val2;
            }
        }else{
            sdata[tid].i = idata[i].i;
            sdata[tid].j = idata[i].j;
            sdata[tid].gain = idata[i].gain;
        }
    }else{
        sdata[tid].i = 0;
        sdata[tid].j = 0;
        sdata[tid].gain = 0;
    }
    __syncthreads();


    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(tid + s < blockDim.x){
            if(sdata[tid].gain < sdata[tid + s].gain){
                sdata[tid].i = sdata[tid + s].i;
                sdata[tid].j = sdata[tid + s].j;
                sdata[tid].gain = sdata[tid + s].gain;
            }
        }
        __syncthreads();
    }

    if(tid == 0){
        odata[blockIdx.x] = sdata[0];
    }
}


__global__ void ILS(uint32_t *indices,
                    int32_t *tour_1,
                    int32_t *tour_2,
                    uint32_t n,
                    hiprandState *states,
                    uint32_t iteration_count,
                    gain_struct_t *buf_1,
                    gain_struct_t *buf_2,
                    uint32_t buffer_size        
                ){
    setup_kernel<<<1,256>>>(states);
    generate_kernel<<<1,256>>>(states, n, indices, iteration_count);
    
    int job = ((n - 1) * (n - 2)) / 2;
    int thread;
    if(job > 1024) thread = 1024;
    else if(job > 512) { thread = 1024; }
    else if(job > 256) { thread = 512;  }
    else if(job > 128) { thread = 256;  }
    else if(job > 64)  { thread = 128;  }
    else if(job > 32)  { thread = 64;   }
    else               { thread = 32;   }

    int block = (job + thread - 1) / thread;
    while(thread * block > buffer_size) --block;

    for(int iter = 0; iter < iteration_count; ++iter){
        perturbate<<<5, 32>>>(tour_1, tour_2, indices[4 * iter], indices[4 * iter + 1], indices[4 * iter + 2], indices[4 * iter + 3], n);
        float perturbation_cost = calculate_perturbate_cost(n, indices[4 * iter], indices[4 * iter + 1], indices[4 * iter + 2], indices[4 * iter + 3], tour_1);
        float gain = 0;
        bool positiveMoveExists;
        do{
            two_opt<<<block, thread, 2 * n * sizeof(int32_t)>>>(tour_2, buf_1, n);
            int order = 0;
            int left = buffer_size;
            while(left > 1){
                int reduce_thread_size = find_num_of_threads((left + 1) / 2, 1024); //1 thread can handle 2 values
                int reduce_block_size = (((left + 1) / 2) + reduce_thread_size - 1) / reduce_thread_size;    
                
                reduce<<<reduce_block_size, reduce_thread_size, sizeof(gain_struct_t) * reduce_thread_size>>>
                ((order == 0 ? buf_1 : buf_2), 
                (order == 0 ? buf_2 : buf_1), left);
                left = reduce_block_size;
                order = 1 - order;
            }
            hipDeviceSynchronize();
            order = 1 - order;
            uint32_t best_i, best_j;
            float best_gain;
            if(order == 0){
                best_i = buf_2[0].i;
                best_j = buf_2[0].j;
                best_gain = buf_2[0].gain;
            }else{
                best_i = buf_1[0].i;
                best_j = buf_1[0].j;
                best_gain = buf_1[0].gain;
            }
            positiveMoveExists = (best_i != 0 || best_j != n - 1) && best_gain > 0.1;
            if(positiveMoveExists) {
                gain += best_gain;
                apply_2opt_move<<<1, 32>>>(tour_2, (order == 0) ? buf_2 : buf_1, n); 
            }
        }while(positiveMoveExists);

        if(gain > perturbation_cost){
            int32_t *temp = tour_1;
            tour_1 = tour_2;
            tour_2 = temp;
        }

    }
}

int main(){
    auto coordinate_city_map = new std::unordered_map<std::pair<int32_t, int32_t>, uint32_t, pair_hash>();
    uint32_t n;
    uint32_t iteration_count = 10000;
    std::ifstream fstream("coordinates.txt");
    if(!fstream.is_open()){
        std::cerr << "coordinates.txt cannot be opened" << std::endl;
        return 1;
    }
    fstream >> n;
    int32_t *h_tour = new int32_t[2 * n];
    uint32_t buffer_size = 4096;
    uint32_t *h_tour_raw = new uint32_t[n];
    gain_struct_t *h_buffer = new gain_struct_t[buffer_size];

    for(int i = 0; i < n; ++i){
        uint32_t city_index;
        int32_t x_coord, y_coord;
        fstream >> city_index; //cities are one based
        fstream >> x_coord;
        fstream >> y_coord;
	h_tour[city_index - 1] = x_coord;
        h_tour[city_index - 1 + n] = y_coord;
	auto pair1 = std::make_pair(x_coord, y_coord);
        if(!coordinate_city_map->insert(std::make_pair(pair1, city_index)).second){
         	std::cout << "i:" << city_index << " x:" << x_coord << " y:" << y_coord  << std::endl;
  	}
	
    }
    fstream.close();

    uint32_t *d_perturbation_indices;
    int32_t *d_tour_1;
    int32_t *d_tour_2;
    gain_struct_t *d_buf_1;
    gain_struct_t *d_buf_2;
    hiprandState *d_curand_states;

    hipMalloc(&d_perturbation_indices, sizeof(uint32_t) * 4 * iteration_count);
    hipMalloc(&d_curand_states, sizeof(hiprandState) * 256);
    hipMalloc(&d_tour_1, sizeof(int32_t) * n * 2);
    hipMalloc(&d_tour_2, sizeof(int32_t) * n * 2);
    hipMalloc(&d_buf_1, sizeof(gain_struct_t) * buffer_size);
    hipMalloc(&d_buf_2, sizeof(gain_struct_t) * buffer_size);
    hipMemcpy(d_buf_1, h_buffer, sizeof(gain_struct_t) * buffer_size, hipMemcpyHostToDevice);
    hipMemcpy(d_buf_2, h_buffer, sizeof(gain_struct_t) * buffer_size, hipMemcpyHostToDevice);

    hipMemcpy(d_tour_1, h_tour, sizeof(int32_t) * n * 2, hipMemcpyHostToDevice);
    //int *sanity = new int[n];
    //for(int i = 0; i < n; ++i) sanity[i] = 1;
        
    //for(int i = 0; i < n; ++i){
    //    int32_t cx = h_tour[i];
    //    int32_t cy = h_tour[i + n];
    //    uint32_t val = coordinate_city_map->at(std::make_pair(cx, cy));
    //    sanity[val - 1] -= 1;
    //}
    
    //for(int i = 0; i < n; ++i) if(sanity[i] != 0) {std::cout << "bad " << i << std::endl; break; }
   

    ILS<<<1,1>>>(d_perturbation_indices,
                 d_tour_1,
                 d_tour_2,
                 n,
                 d_curand_states,
                 iteration_count,
                 d_buf_1,
                 d_buf_2,
                 buffer_size);

    std::ofstream par("par.txt");
    hipMemcpy(h_tour, d_tour_1, sizeof(int32_t) * 2 * n, hipMemcpyDeviceToHost);
    for(int i = 0; i < n; ++i){
        int32_t cx = h_tour[i];
        int32_t cy = h_tour[i + n];
        h_tour_raw[i] = coordinate_city_map->at(std::make_pair(cx, cy));
    	par << h_tour_raw[i] << " ";
    }
    par << std::endl;
    par.close();
                 
    hipFree(d_perturbation_indices);
    hipFree(d_curand_states);
    hipFree(d_tour_1);
    hipFree(d_tour_2);
    hipFree(d_buf_1);
    hipFree(d_buf_2);
    delete[] h_tour;
    delete[] h_buffer;
    delete[] h_tour_raw;
}
